
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024
#define BLOCK_SIZE 16
#define NUM_STREAMS 4
 
__global__ void matrix_multiply(float *a, float *b, float *c, int start_idx, int end_idx) {
    for (int idx = start_idx; idx < end_idx; idx++) {
        for (int i = 0; i < N; i++) {
            float sum = 0.0;
            for (int k = 0; k < N; k++) {
                sum += a[i * N + k] * b[k * N + idx];
            }
            c[i * N + idx] = sum;
        }
    }
}

void matrix_multiply_task(float *a, float *b, float *c, int start_idx, int end_idx, hipStream_t stream) {
    float *d_a, *d_b, *d_c;
    int size = N * (end_idx - start_idx) * sizeof(float);

    hipMalloc((void**) &d_a, N * N * sizeof(float));
    hipMalloc((void**) &d_b, N * (end_idx - start_idx) * sizeof(float));
    hipMalloc((void**) &d_c, N * (end_idx - start_idx) * sizeof(float));

    hipMemcpyAsync(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b + start_idx * N, size, hipMemcpyHostToDevice, stream);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (end_idx - start_idx + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_multiply<<<dimGrid, dimBlock, 0, stream>>>(d_a, d_b, d_c, start_idx, end_idx);

    hipMemcpyAsync(c + start_idx * N, d_c, size, hipMemcpyDeviceToHost, stream);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    float *a, *b, *c;
    hipStream_t streams[NUM_STREAMS];

    a = (float*) malloc(N * N * sizeof(float));
    b = (float*) malloc(N * N * sizeof(float));
    c = (float*) calloc(N * N, sizeof(float));

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Initialize matrices a and b with some values

    int chunk_size = N / NUM_STREAMS;

    for (int i = 0; i < NUM_STREAMS; i++) {
        int start_idx = i * chunk_size;
        int end_idx = (i + 1) * chunk_size;
        if (i == NUM_STREAMS - 1) {
            end_idx = N;
        }
        matrix_multiply_task(a, b, c, start_idx, end_idx, streams[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    // Print matrix c

    free(a);
    free(b);
    free(c);

    return 0;
}
